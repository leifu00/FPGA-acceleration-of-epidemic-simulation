
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <vector>
#include <iostream>


using namespace std;

int problem_size = 20000000;


float random_float(float a, float b)
{
    float random = ((float) rand()) / (float) RAND_MAX;
    float diff = b - a;
    float r = random * diff;
    return a + r;
}

int random_int(int min, int max)    
{    
    return rand() % (max - min) + min + 1;     
}


__global__
void infect_sweep(int *InfStats, int *Travelling, float *HouseInf, int *Absent, float *WAIFW_Matrix, float *AgeSusceptibility, 
        int *Age, float *Susceptibility, int *Treated, int *Vaccinated, float *Results)
{
    int i =  (blockIdx.x * blockDim.x) + threadIdx.x;
    float FOI = HouseInf[i];

    for (int j = i * 10; j < (i + 1) * 10; j++)
    {
        FOI *= (1 + 2 * Absent[j]);
    }
        
    for (int j = i * 10; j < (i + 1) * 10; j++)
    {
        int host_age_group = Age[j] / 17;
        int infector_age_group = Age[i] / 17;
        float infectee_susceptibility = AgeSusceptibility[host_age_group] * Susceptibility[j];
        FOI *= WAIFW_Matrix[host_age_group * 17 + infector_age_group] * infectee_susceptibility;
        FOI *= (1.0 - Treated[j] * 2) * (1.0 - Vaccinated[j] * 2) * Travelling[i] * InfStats[i];
        Results[j] = FOI;
    }
}

int main(void)
{

        int *InfStats_h = (int *)malloc(problem_size*sizeof(int));
        int *Travelling_h = (int *)malloc(problem_size*sizeof(int));
        int *HouseInf_h = (int *)malloc(problem_size*sizeof(float));
    
        float *WAIFW_Matrix_h = (float *)malloc(100*sizeof(float));
        float *AgeSusceptibility_h = (float *)malloc(100*sizeof(float));
    
        int *Absent_h = (int *)malloc(15*problem_size*sizeof(int));
        int *Age_h = (int *)malloc(15*problem_size*sizeof(int));
        int *Treated_h = (int *)malloc(15*problem_size*sizeof(int));
        int *Vaccinated_h = (int *)malloc(15*problem_size*sizeof(int));
    
        int *Susceptibility_h = (int *)malloc(15*problem_size*sizeof(float));    
        int *Results_h = (int *)malloc(15*problem_size*sizeof(float));    
    
    
        {
            for (int i = 0; i < problem_size; i++)
            {
                InfStats_h[i] = 1;
                Travelling_h[i] = 0;
                HouseInf_h[i] = random_float(0.1, 1);
            }
        
            for (int i = 0; i < 100; i++)
            {
                AgeSusceptibility_h[i] = random_float(0.1, 1);
                WAIFW_Matrix_h[i] = random_float(0.1, 1);
            }
        
            for (int i = 0; i < problem_size * 10; i++)
            {
                Age_h[i] = random_int(0, 100);
                Susceptibility_h[i] = random_float(0.1, 1);
                Treated_h[i] =  0;
                Vaccinated_h[i] = 1;
                Results_h[i] = 0;
            }
         
        }
    
     
    
      
        int *InfStats_d, *Travelling_d, *Absent_d, *Age_d, *Treated_d, *Vaccinated_d;
        float *HouseInf_d,  *WAIFW_Matrix_d, *AgeSusceptibility_d, *Susceptibility_d, *Results_d;
    
        hipMalloc(&InfStats_d, problem_size*sizeof(int)); 
        hipMalloc(&Travelling_d, problem_size*sizeof(int)); 
        hipMalloc(&Absent_d, 15*problem_size*sizeof(int)); 
        hipMalloc(&Age_d, 15*problem_size*sizeof(int)); 
        hipMalloc(&Treated_d, 15*problem_size*sizeof(int)); 
        hipMalloc(&Vaccinated_d, 15*problem_size*sizeof(int)); 
    
        hipMalloc(&HouseInf_d, problem_size*sizeof(float)); 
        hipMalloc(&WAIFW_Matrix_d, 100*sizeof(float)); 
        hipMalloc(&AgeSusceptibility_d, 100*sizeof(float)); 
        hipMalloc(&Susceptibility_d, 15*problem_size*sizeof(float)); 
        hipMalloc(&Results_d, 15*problem_size*sizeof(float)); 
    
        struct timeval t1, t2;
        gettimeofday(&t1, 0);
    
        hipMemcpy(InfStats_d, InfStats_h, problem_size*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(Travelling_d, Travelling_h, problem_size*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(Absent_d, Absent_h, problem_size*sizeof(int)*15, hipMemcpyHostToDevice);
        hipMemcpy(Age_d, Age_h, problem_size*sizeof(int)*15, hipMemcpyHostToDevice);
        hipMemcpy(Treated_d, Treated_h, problem_size*sizeof(int)*15, hipMemcpyHostToDevice);
        hipMemcpy(Vaccinated_d, Vaccinated_h, problem_size*sizeof(int)*15, hipMemcpyHostToDevice);
        hipMemcpy(HouseInf_d, HouseInf_h, problem_size*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(WAIFW_Matrix_d, WAIFW_Matrix_h, 100*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(AgeSusceptibility_d, AgeSusceptibility_h, 100*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(Susceptibility_d, Susceptibility_h, problem_size*sizeof(float)*15, hipMemcpyHostToDevice);
        hipMemcpy(Results_d, Results_h, problem_size*sizeof(float)*15, hipMemcpyHostToDevice);
    
    
      infect_sweep<<<(problem_size+255)/256, 256>>>(InfStats_d, Travelling_d, HouseInf_d, Absent_d, WAIFW_Matrix_d, AgeSusceptibility_d, 
        Age_d, Susceptibility_d, Treated_d, Vaccinated_d, Results_d);
    
      hipMemcpy(Results_h, Results_d, problem_size*15*sizeof(float), hipMemcpyDeviceToHost);
      gettimeofday(&t2, 0);
      double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
      printf("Population size: %d. Time to generate:  %3.1f ms.\n", (problem_size*10), time);
    
      free(InfStats_h);
      free(Travelling_h);
      free(HouseInf_h);
      free(WAIFW_Matrix_h);
      free(AgeSusceptibility_h);
      free(Absent_h);
      free(Age_h);
      free(Treated_h);
      free(Vaccinated_h);
      free(Susceptibility_h);
      free(Results_h);
    
      hipFree(InfStats_d);
      hipFree(Travelling_d);
      hipFree(Absent_d);
      hipFree(Age_d);
      hipFree(Treated_d);
      hipFree(Vaccinated_d);
      hipFree(InfStats_d);
    
      hipFree(HouseInf_d);
      hipFree(WAIFW_Matrix_d);
      hipFree(AgeSusceptibility_d);
      hipFree(Susceptibility_d);
      hipFree(Results_d);
   

}